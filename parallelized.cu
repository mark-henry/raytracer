#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "Image.h"
#include "types.h"
#include "vector_math.h"
#include <math.h>

#define NUM_SPHERES 100
#define SPHERE_RADIUS 1
#define NUM_LIGHTS 1
#define DRAW_DIST 50
#define SCENE_SEED 3

#define IMG_WIDTH 1024
#define IMG_HEIGHT 1024
#define BLOCK_DIM 32

void generateScene(sphere_t *spheres, point_light_t *lights, camera_t *camera)
{
   // Generates NUM_SPHERES spheres, randomly placed in the frustum of the camera
   //  from DRAW_DIST/4 out to DRAW_DIST

   // The camera is at the origin looking along +Z
   camera->position = (vector_t){0,0,-2};
   camera->look = (vector_t){0,0,1};
   camera->up = (vector_t){0,1,0};
   
   srand(SCENE_SEED);

   vector_t pos;
   material_t mat;
   double spread = 1.5;
   int zval;
   for (int i = 0; i < NUM_SPHERES; i++)
   {
      zval = DRAW_DIST - pow(rand() % (int)sqrt(DRAW_DIST), 2);
      pos.z = zval;
      pos.x = (rand() % (zval+1) - (zval / 2)) * spread;
      pos.y = (rand() % (zval+1) - (zval / 2)) * spread;

      mat.diffuse.r = (rand() % 100) / 100.0;
      mat.diffuse.g = (rand() % 100) / 100.0;
      mat.diffuse.b = (rand() % 100) / 100.0;
      
      mat.specular.r = (rand() % 100) / 100.0;
      mat.specular.g = (rand() % 100) / 100.0;
      mat.specular.b = (rand() % 100) / 100.0;
      
      mat.ambient = (color_t){.05, .05, .05};
      
      mat.shininess = pow((rand() % 9) + 2, 2);

      spheres[i].position = pos;
      spheres[i].radius = 1;
      spheres[i].material = mat;
   }

   // Place a single light in the middle of the spheres
   lights[0].position = (vector_t){0,0,DRAW_DIST/3};
   lights[0].color = (color_t){1,1,1};
}

// Set up rays based on camera position and image size
void initRays(ray_t *rays, camera_t camera,
              int img_height, int img_width)
{
   ray_t ray;
   double aspectRatio = (double)img_height / img_width;
   vector_t rightShift, upShift;
   double u, v;
   vector_t right;

   normalize(&camera.look);
   normalize(&camera.up);
   right = cross(camera.look, camera.up);
   
   // Iterate over all pixels
   for (int y = 0; y < img_height; y++) {
      for (int x = 0; x < img_width; x++)
      {
         // Calculate ray direction
         u = aspectRatio * x / img_width * 2 - 1;
         v = -((double)y / img_height * 2 - 1);
         rightShift = multiply(right, u);
         upShift = multiply(camera.up, v);

         ray.start = camera.position;
         ray.dir = add(camera.look, add(rightShift, upShift));

         // Which pixel in the array does this ray correspond to?
         ray.pixel = y*img_width + x;
         rays[y*img_width + x] = ray;
      }
   }
}

void writeImage(char *filename, color_t *image, int width, int height)
{
   Image img(width, height);

   // Copy image to Image object
   // Image is weird: (0,0) is the lower left corner
   for (int y = 0; y < height; y++)
      for (int x = 0; x < width; x++)
         img.pixel(x, height-y-1, image[width*y + x]);
   
   img.WriteTga(filename, false);
}

// Returns the t-parameter value of the intersection between
//  a ray and a sphere.
// Returns a negative value if the ray misses the sphere.
__device__ double sphereIntersectionTest(sphere_t *sphere, ray_t *in_ray)
{
   // For explanation of algorithm, see http://tinyurl.com/yjoup3w
   
   // Transform ray into sphere space
   ray_t ray = *in_ray;
   ray.start = subtract(ray.start, sphere->position);

   // We must solve the quadratic equation with A, B, C equal to:
   double A = dotProduct(ray.dir, ray.dir);
   double B = 2*dotProduct(ray.dir, ray.start);
   double C = dotProduct(ray.start, ray.start) -
               sphere->radius * sphere->radius;

   // If the discriminant is negative, the ray has missed the sphere
   double discriminant = B*B - 4*A*C;
   if (discriminant < 0)
      return discriminant;

   // q is an intermediary value in finding the solutions
   double q;
   if (B < 0)
      q = (-B - sqrtf(discriminant))/2.0;
   else
      q = (-B + sqrtf(discriminant))/2.0;

   // Compute the t-values of the intersections
   double t0 = q / A;
   double t1 = C / q;

   // Do a little branch just in case the camera is inside the sphere
   if (t0 > 0 && t1 > 0)
      return min(t0, t1);
   else
      return max(t0, t1);
}

// Helper function for illumination calculations
inline __device__ void addLightingFactor(color_t *illum, color_t material, color_t light)
{
   illum->r += material.r * light.r;
   illum->g += material.g * light.g;
   illum->b += material.b * light.b;
}

// Calculates the color of a ray which is known to intersect a sphere
__device__ color_t directIllumination(sphere_t *sphere, ray_t *ray, double t,
                                      point_light_t *lights, int num_lights)
{
   color_t illum = {0,0,0};

   // inter is the position of the intersection point
   vector_t inter = add(ray->start, multiply(ray->dir, t));

   // normal is the surface normal at the point of intersection
   vector_t normal = subtract(inter, sphere->position);
   normalize(&normal);

   // V is the eye vector
   vector_t V = ray->dir;
   normalize(&V);

   // Add diffuse and specular for each point_light
   for (int li = 0; li < num_lights; li++) {
      // L is the incident light vector
      vector_t L = subtract(lights[li].position, inter);
      normalize(&L);

      // Add ambient
      addLightingFactor(&illum, sphere->material.ambient, lights[li].color);

      // Add diffuse
      double dotProd = max(0.0, dotProduct(normal, L));
      addLightingFactor(&illum, multiply(sphere->material.diffuse, dotProd),
                        lights[li].color);

      // Add specular
      vector_t R = reflection(L, normal);
      double specDotProd = pow(min(0.0, dotProduct(V, R)), sphere->material.shininess);
      addLightingFactor(&illum, multiply(sphere->material.specular, specDotProd),
                        lights[li].color);
   }

   illum.r = min(illum.r, 1.0);
   illum.g = min(illum.g, 1.0);
   illum.b = min(illum.b, 1.0);
   return illum;
}

// Finds the color of an arbitrary ray
__device__ color_t castRay(ray_t *ray,
                           sphere_t *spheres, int num_spheres,
                           point_light_t *lights, int num_lights)
{
   color_t bgColor = {0,0,0};
   color_t rayColor = bgColor;

   // Does this ray intersect with any spheres?
   double closest = DRAW_DIST;
   double t;
   for (int sphere = 0; sphere < num_spheres; sphere++) {
      t = sphereIntersectionTest(&spheres[sphere], ray);
      if (t > 0 && t < closest) {
         closest = t;
         rayColor = directIllumination(&spheres[sphere], ray, t, lights, num_lights);
      }
   }

   return rayColor;
}

// Takes in a scene and outputs an image
__global__ void rayTrace(ray_t *rays, int num_rays,
                         sphere_t *spheres, int num_spheres,
                         point_light_t *lights, int num_lights,
                         color_t *pixels, int num_pixels)
{

   int rayIdx = blockDim.x * blockIdx.x + threadIdx.x;
   pixels[rays[rayIdx].pixel] =
      castRay(&rays[rayIdx], spheres, num_spheres, lights, num_lights);
}

int main(void)
{
   sphere_t *spheres, *dev_spheres;
   point_light_t *lights, *dev_lights;
   ray_t *rays, *dev_rays;
   color_t *image, *dev_image;
   camera_t camera;

   int spheres_size = NUM_SPHERES * sizeof(sphere_t);
   int lights_size  = NUM_LIGHTS * sizeof(point_light_t);
   int rays_size  = IMG_HEIGHT*IMG_WIDTH*sizeof(ray_t);
   int image_size = IMG_HEIGHT*IMG_WIDTH*sizeof(color_t);
   
   spheres = (sphere_t *)      malloc(spheres_size);
   lights  = (point_light_t *) malloc(lights_size);
   rays    = (ray_t *)         malloc(rays_size);
   image   = (color_t *)       malloc(image_size);
   
   generateScene(spheres, lights, &camera);

   initRays(rays, camera, IMG_WIDTH, IMG_HEIGHT);

   // hipMalloc dev_ arrays
   hipMalloc(&dev_spheres, spheres_size);
   hipMalloc(&dev_lights, lights_size);
   hipMalloc(&dev_rays, rays_size);
   hipMalloc(&dev_image, image_size);
   
   // hipMemcpy the problem to device
   hipMemcpy(dev_spheres, spheres, spheres_size, hipMemcpyHostToDevice);
   hipMemcpy(dev_lights, lights, lights_size, hipMemcpyHostToDevice);
   hipMemcpy(dev_rays, rays, rays_size, hipMemcpyHostToDevice);
   
   // Invoke kernel
   int dimBlock = BLOCK_DIM;
   int dimGrid = (IMG_HEIGHT*IMG_WIDTH) / BLOCK_DIM;
   rayTrace<<<dimGrid, dimBlock>>>(dev_rays, IMG_HEIGHT*IMG_WIDTH,
                                   dev_spheres, NUM_SPHERES,
                                   dev_lights, NUM_LIGHTS,
                                   dev_image, IMG_HEIGHT*IMG_WIDTH);

   // hipMemcpy the result image from the device
   hipMemcpy(image, dev_image, image_size, hipMemcpyDeviceToHost);
   
   // write image to output file
   writeImage("awesome.tga", image, IMG_WIDTH, IMG_HEIGHT);

   // Free memory
   free(spheres);
   free(lights);
   free(rays);
   free(image);
   hipFree(dev_spheres);
   hipFree(dev_lights);
   hipFree(dev_rays);
   hipFree(dev_image);

   return 0;
}